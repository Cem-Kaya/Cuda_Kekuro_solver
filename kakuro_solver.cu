#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>
#include <array>
//#include <bits/stdc++.h>


#include <hip/hip_runtime.h>
using namespace std;

enum direction { d_down, d_right, none };

#define COORD std::pair<int, int>

//#define DEBUG

int iter = 0;

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int* arr, int n) {

	cout << "arr: ";

	for (int i = 0; i < n; i++) {
		cout << arr[i] << " ";
	}

	cout << endl;

}

void print_coords(COORD start, COORD end) {

	cout << "Start:" << start.first << "," << start.second << endl;
	cout << "End:" << end.first << "," << end.second << endl;

}

int find_length(COORD start, COORD end, direction dir) {

	if (dir == d_down)
		return end.first - start.first;
	if (dir == d_right)
		return end.second - start.second;

	return -1;
}

void convert_sol(int** mat, int**& sol_mat, int m, int n) {

	sol_mat = new int* [m]; //Rows
	for (int i = 0; i < m; i++) {
		sol_mat[i] = new int[n]; //Cols
	}

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			if (mat[i][j] == -2)
				sol_mat[i][j] = -2; //Empty value cell
			else
				sol_mat[i][j] = -1; //Hint or empty cell
		}
	}
}

void print_one_matrix(int** matrix, int m, int n) {
	std::cout << "Matrix: " << std::endl;
	for (int i = 0; i < m; i++) { //rows
		for (int j = 0; j < n; j++) { //cols
			std::cout << matrix[i][j] << "\t";
		}
		std::cout << "\n";
	}
}

///Auxiliary functions

struct sum {
	COORD start;
	COORD end;

	int hint;
	int dir;
	int length;
	int* arr;

	void print_sum() {
		cout << "############################" << endl;
		cout << "Creating sum with: " << endl;
		print_coords(start, end);
		cout << "Hint: " << hint << endl;
		cout << "Direction: " << dir << endl;
		cout << "Length: " << length << endl;
		cout << "############################" << endl;
	}

	sum(COORD _start, COORD _end, int _hint, direction _dir) :
		start(_start), end(_end), hint(_hint), dir(_dir)
	{
		length = find_length(_start, _end, _dir);
		arr = new int[length];
#ifdef DEBUG
		cout << "############################" << endl;
		cout << "Creating sum with: " << endl;
		print_coords(start, end);
		cout << "Hint: " << hint << endl;
		cout << "Direction: " << dir << endl;
		cout << "Length: " << length << endl;
		cout << "############################" << endl;
#endif
	}

	//~sum(){
	//delete arr;
	//}
};


COORD find_end(int** matrix, int m, int n, int i, int j, direction dir) { //0 down 1 right

	if (dir == d_right) {
		for (int jj = j + 1; jj < n; jj++) {
			if (matrix[i][jj] != -2 || jj == n - 1) {
				if (matrix[i][jj] == -2 && jj == n - 1)
					jj++;
				COORD END = COORD(i, jj);
				return END;
			}
		}
	}

	if (dir == d_down) {
		for (int ii = i + 1; ii < m; ii++) {
			if (matrix[ii][j] != -2 || ii == m - 1) {
				if (matrix[ii][j] == -2 && ii == m - 1)
					ii++;
				COORD END = COORD(ii, j);
				return END;
			}
		}
	}

}


vector<sum> get_sums(int** matrix, int m, int n) {

	vector<sum> sums;

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			int val = matrix[i][j];
			if (val != -1 && val != -2) {
				int hint = val;
				hint = hint / 10;

				if ((hint % 100) == 0) {
					hint = (int)(hint / 100);
					COORD START = COORD(i, j + 1);
					COORD END = find_end(matrix, m, n, i, j, d_right);
					sum _sum = sum(START, END, hint, d_right);
					sums.push_back(_sum);
				}

				else {
					int div = (int)(hint / 100);
					int rem = (int)(hint % 100);

					if (div == 0 && rem != 0) {
						COORD START = COORD(i + 1, j);
						COORD END = find_end(matrix, m, n, i, j, d_down);
						sum _sum = sum(START, END, rem, d_down);
						sums.push_back(_sum);
					}

					if (div != 0 && rem != 0) {
						COORD START1 = COORD(i + 1, j);
						COORD START2 = COORD(i, j + 1);
						COORD END1 = find_end(matrix, m, n, i, j, d_down);
						COORD END2 = find_end(matrix, m, n, i, j, d_right);
						sum _sum1 = sum(START1, END1, rem, d_down);
						sum _sum2 = sum(START2, END2, div, d_right);
						sums.push_back(_sum1);
						sums.push_back(_sum2);
					}
				}
			}


		}
	}
	return sums;
}


void read_matrix(int**& matrix, std::ifstream& afile, int m, int n) {

	matrix = new int* [m]; //rows

	for (int i = 0; i < m; i++) {
		matrix[i] = new int[n]; //cols
	}

	int val;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			afile >> val;
			matrix[i][j] = val;
		}
	}
}

void sol_to_file(int** mat, int** sol_mat, int m, int n) {

	string fname = "visualize.kakuro";
	ofstream to_write(fname);

	to_write << m << " " << n << "\n";

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			if (mat[i][j] != -2)
				to_write << mat[i][j] << " ";
			else
				to_write << sol_mat[i][j] << " ";
		}
		to_write << "\n";
	}

	to_write.close();
}

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums) {

	for (int i = 0; i < no_sums; i++) {

		h_sum_starts_x[i] = sums[i].start.first;
		h_sum_starts_y[i] = sums[i].start.second;

		h_sum_ends_x[i] = sums[i].end.first;
		h_sum_ends_y[i] = sums[i].end.second;

		h_sum_hints[i] = sums[i].hint;
		h_sum_lengths[i] = sums[i].length;

		h_sum_dirs[i] = sums[i].dir;
	}

}

void print_flattened(int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums) {

	cout << "###h_sum_starts_x: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_starts_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_starts_y: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_starts_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_x: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_ends_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_y: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_ends_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_hints: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_hints[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_lengths: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_lengths[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_dirs: " << endl;
	for (int i = 0; i < no_sums; i++) {
		cout << h_sum_dirs[i] << " ";
	}
	cout << endl;

}

void flatten_sol_mat(int** sol_mat, int* h_sol_mat, int m, int n) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			h_sol_mat[i * n + j] = sol_mat[i][j];
		}
	}
}

void print_flattened_matrix(int* h_sol_mat, int m, int n) {

	cout << "###Flattened matrix: " << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			cout << h_sol_mat[i * n + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////


///////////////////
//CUDA FUNCTIONS //
///////////////////
__device__ void d_print_flattened_matrix(int* d_sol_mat, int m, int n) {
	printf("_device_ matrix: %d %d \n", m, n);
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			printf("%d ", d_sol_mat[i * n + j]);
		}
		printf("\n");
	}
	printf("\n");
}

__device__ int get_cord_to_change(int& current_ind, int last_index, int* mat_arr) {
	while (current_ind < last_index) {
		if (mat_arr[current_ind] == -2) {
			return current_ind;
		}
		current_ind++;
	}
	return -10; // why ? i dont know 
}

__device__ void copy_mat(int* from, int* to, int last_index) {
	for (int i = 0; i < last_index; i++) {
		to[i] = from[i];
	}
}

struct state_data {
	int x_cord;
	int y_cord;
	int val;
	__device__ state_data() : x_cord(-10), y_cord(-10), val(-10) {}
	__device__ state_data(int x, int y, int v) : x_cord(x), y_cord(y), val(v) {}
};

struct my_gpu_stack {
	int top;
	int maxSize;
	state_data* data;

	__device__ my_gpu_stack(int max) : top(-1), maxSize(max*10) {
		data = new state_data[maxSize];
	}

	__device__ ~my_gpu_stack() {
		delete[] data;
	}

	__device__ void push(state_data value) {
		if (top < maxSize - 1) {
			data[++top] = value;
		}
		else {
			printf("Stack Overflow\n");
		}
	}

	__device__ state_data pop() {
		if (top >= 0) {
			return data[top--];
		}
		else {
			printf("Stack Underflow\n");
			return state_data();
		}
	}

	__device__ bool is_empty() {
		return top == -1;
	}
};

__device__ bool check_a_spesific_sum_if_contains_duplicates(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int spesific_sum_index, int m, int n) {
	int start_x = d_sum_starts_x[spesific_sum_index];
	int start_y = d_sum_starts_y[spesific_sum_index];
	int end_x = d_sum_ends_x[spesific_sum_index];
	int end_y = d_sum_ends_y[spesific_sum_index];
	int hint = d_sum_hints[spesific_sum_index];
	int length = d_sum_lengths[spesific_sum_index];
	int dir = d_sum_dirs[spesific_sum_index];
	bool has_duplicates = false;
	if (dir == 0) {
		for (int j = start_x; j <= end_x; j++) {
			for (int k = j + 1; k <= end_x; k++) {
				if (d_sol_mat[j * n + start_y] == d_sol_mat[k * n + start_y]) {
					has_duplicates = true;
					return has_duplicates;
				}
			}
		}
	}
	else {
		for (int j = start_y; j <= end_y; j++) {
			for (int k = j + 1; k <= end_y; k++) {
				if (d_sol_mat[start_x * n + j] == d_sol_mat[start_x * n + k]) {
					has_duplicates = true;
					return has_duplicates;
				}
			}
		}
	}
	return has_duplicates;
}

__device__ bool check_a_spesific_sum_if_contains_2(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int spesific_sum_index, int m, int n) {
	int start_x = d_sum_starts_x[spesific_sum_index];
	int start_y = d_sum_starts_y[spesific_sum_index];
	int end_x = d_sum_ends_x[spesific_sum_index];
	int end_y = d_sum_ends_y[spesific_sum_index];
	int hint = d_sum_hints[spesific_sum_index];
	int length = d_sum_lengths[spesific_sum_index];
	int dir = d_sum_dirs[spesific_sum_index];
	bool does_contain_2 = false;
	if (dir == 0) {
		for (int j = start_x; j <= end_x; j++) {
			if (d_sol_mat[j * n + start_y] == -2) {
				does_contain_2 = true;
				return does_contain_2;
			}
		}
	}
	else {
		for (int j = start_y; j <= end_y; j++) {
			if (d_sol_mat[start_x * n + j] == -2) {
				does_contain_2 = true;
				return does_contain_2;
			}
		}
	}
	return does_contain_2;
}


__device__ int sum_a_spesific_sum(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int spesific_sum_index, int m, int n) {
	int start_x = d_sum_starts_x[spesific_sum_index];
	int start_y = d_sum_starts_y[spesific_sum_index];
	int end_x = d_sum_ends_x[spesific_sum_index];
	int end_y = d_sum_ends_y[spesific_sum_index];
	int hint = d_sum_hints[spesific_sum_index];
	int length = d_sum_lengths[spesific_sum_index];
	int dir = d_sum_dirs[spesific_sum_index];
	int sum_of_sum = 0;
	if (dir == 0) {
		for (int j = start_x; j <= end_x; j++) {
			sum_of_sum += d_sol_mat[j * n + start_y];
		}
	}
	else {
		for (int j = start_y; j <= end_y; j++) {
			sum_of_sum += d_sol_mat[start_x * n + j];
		}
	}
	return sum_of_sum;
}

__device__ bool check_singe_sum_is_correct(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int spesific_sum_index, int m, int n) {
	int sum_of_sum = sum_a_spesific_sum(d_sol_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, spesific_sum_index, m, n);
	int hint = d_sum_hints[spesific_sum_index];
	if (sum_of_sum != hint) {
		return false;
	}
	else {
		if (check_a_spesific_sum_if_contains_2(d_sol_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, spesific_sum_index, m, n)) {
			return false;
		}
		else {
			if (check_a_spesific_sum_if_contains_duplicates(d_sol_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, spesific_sum_index, m, n)) {
				return false;
			}
			else {
				return true;
			}
		}
	}
}

__device__ bool check_singe_sum_is_Viable(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int spesific_sum_index, int m, int n) {
	int sum_of_sum = sum_a_spesific_sum(d_sol_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, spesific_sum_index, m, n);
	int hint = d_sum_hints[spesific_sum_index];
	if (sum_of_sum <= hint) {
		return true;
	}
	else {
		return false;
	}
}

__device__ bool check_all_the_sums_are_correct(int* d_sol_mat, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int m, int n) {
	for (int i = 0; i < no_sums; i++) {
		if (!check_singe_sum_is_correct(d_sol_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, i, m, n)) {
			return false;
		}
	}
	return true;
}

// enum direction { d_down, d_right, none };
// for a given kakuro board make a lookup table which takes cordinate x,y and returns the two 
__device__ int* make_lookup_table(int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, int no_sums, int m, int n) {
	int* lookup_table = new int[m * n * 2];
	for (int i = 0; i < m * n * 2; i++) {
		lookup_table[i] = -1;
	}
	for (int i = 0; i < no_sums; i++) {
		int start_x = d_sum_starts_x[i];
		int start_y = d_sum_starts_y[i];
		int end_x = d_sum_ends_x[i];
		int end_y = d_sum_ends_y[i];
		int dir = d_sum_dirs[i];
		if (dir == 1) { // if direction is down
			for (int j = start_y; j < end_y ; j++) {
				int index = (start_x * n + j) ;
				if (lookup_table[index] == -1) {
					lookup_table[index] = i;
				}
				else {
					lookup_table[index + m * n] = i; // tune this 
				}
			}
		}
		else { // if direction is right
			for (int j = start_x; j < end_x ; j++) {
				int index = (j * n + start_y);
				if (lookup_table[index] == -1) {
					lookup_table[index] = i;
				}
				else {
					lookup_table[index + m*n ] = i;
				}
			}
		}

	}
	return lookup_table;
}


__global__ void kakuro_kernel(
	int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs,
	int* d_sol_mat, int* d_perms,
	int* d_t_mats, int m, int n,
	int no_sums, volatile bool* solved) {
	//TO DO
	const int MAT_SIZE = m * n * sizeof(int);
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	int* this_threads_mat = d_t_mats + thread_id * MAT_SIZE;
	my_gpu_stack stack(no_sums);
	copy_mat(d_sol_mat, this_threads_mat, MAT_SIZE / sizeof(int));
	__syncthreads();

	for (int i = 0; i < no_sums; i++) {
		printf(" hints : %d \n", d_sum_hints[i]);
	}

	d_print_flattened_matrix(this_threads_mat, m, n);
	int cur_ind = 0;


	// Initial state

	int tmp = get_cord_to_change(cur_ind, MAT_SIZE / sizeof(int), this_threads_mat);

	int i = 1;
	stack.push(state_data(tmp / m, tmp % n, i));
	
	printf("Hints :");
	for (int i = 0; i < no_sums; i++) {
		printf(" %d ", d_sum_hints[i]);
	}

	int* cord_to_sum_lookup = make_lookup_table(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, m, n);

	printf("\n TABLe \n");	
	d_print_flattened_matrix(cord_to_sum_lookup, m, n);
	printf("\n TABLe2 \n");
	d_print_flattened_matrix(cord_to_sum_lookup + m * n, m, n);
	printf("\n TABLe \n");
	
	while (!stack.is_empty()) {
		d_print_flattened_matrix(this_threads_mat, m, n);


 		int tttmp = 0;
		int is_there_empthy_slot = get_cord_to_change(tttmp, MAT_SIZE / sizeof(int), this_threads_mat);
		
		// is at leaf node ?? 
		if (is_there_empthy_slot == -10) {						
			// TODO do stuff to check if solved and if not give the correct value to the stack 
			if (check_all_the_sums_are_correct(this_threads_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, m, n)) {
				*solved = true;
				return;
			}
			else {
				// TODO add viablitiy check here 
				state_data cur = stack.pop();
				i = cur.val;
				if (i < 10) {
					this_threads_mat[cur.x_cord * m + cur.y_cord] = i;
					i++; // next iterations value
					stack.push(state_data(cur.x_cord, cur.y_cord, i));
				}
				else {
					this_threads_mat[cur.x_cord * m + cur.y_cord] = -2;					
				}				
			} 
		}		
		else {
			state_data cur = stack.pop();
			i = cur.val;
			if (cur.val < 10) {
				this_threads_mat[cur.x_cord * m + cur.y_cord] = i;
				i++; // next iterations value

				// Pruning condition: Check if the related sums are still valid
				bool Viable = false;
				// get inpacted sums and check if they are valid
				int first_effected_sum_index = cord_to_sum_lookup[cur.x_cord * m + cur.y_cord];
				int second_effected_sum_index = cord_to_sum_lookup[cur.x_cord * m + cur.y_cord + m * n];
				printf(" %d  %d \n ", first_effected_sum_index, second_effected_sum_index);
				if (check_singe_sum_is_Viable(this_threads_mat, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, no_sums, first_effected_sum_index, m, n) 
					&& check_singe_sum_is_Viable(this_threads_mat, d_sum_starts_x,  d_sum_starts_y, d_sum_ends_x,  d_sum_ends_y,  d_sum_hints, d_sum_lengths,d_sum_dirs, no_sums, second_effected_sum_index,  m,  n) )
				{
					Viable = true;
				}

				if (Viable) {
					// Save the current state and push it back to the stack
					stack.push(state_data(cur.x_cord, cur.y_cord, i));

					int tttmp = 0;
					// Move on to the next state
					tmp = get_cord_to_change(tttmp, MAT_SIZE / sizeof(int), this_threads_mat);
					//next_cord = get<0>(tmp);
					//sums_index = get<1>(tmp);
					i = 1;
					stack.push(state_data(tmp / m, tmp % n, i));;
				}
				else {
					// Reset the current cell when backtracking
					this_threads_mat[cur.x_cord * m + cur.y_cord] = -2;
					//stack.push(state_data(next_cord, sums_index, i));
					stack.push(state_data(cur.x_cord, cur.y_cord, i));;

				}
			}
			else {
				// Reset the cell to its initial value when backtracking
				this_threads_mat[cur.x_cord * m + cur.y_cord] = -2;
			}
		}
		////////////















		//About volatile bool* solved:
		//You can get idea from https://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does%5B/url%5D for how to break out of a CUDA kernel
		//You may or may not use it

	}
	

}
///////////////////
//CUDA FUNCTIONS //
///////////////////

int main(int argc, char** argv) {

	std::string filename(argv[1]);
	std::ifstream file;
	file.open(filename.c_str());

	int m, n;
	file >> m;
	file >> n;

	int** mat;
	read_matrix(mat, file, m, n);
	print_one_matrix(mat, m, n);

	int** sol_mat;
	convert_sol(mat, sol_mat, m, n);
	//print_one_matrix(sol_mat, m, n);

	vector<sum> sums = get_sums(mat, m, n);

	//CUDA
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
	printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
	printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);
	printf("==prop== Shared Memory Per Block: -- %zu bytes\n", prop.sharedMemPerBlock);
	printf("==prop== Total global memory: -- %zu bytes\n", prop.totalGlobalMem);
	printf("==prop== Clock rate: -- %d\n", prop.clockRate);
	printf("==prop== Compute capability: -- %d.%d\n", prop.major, prop.minor);



	int grid_dim = 1;//TO DO
	int block_dim = 1;//To DO

	int no_sums = sums.size();


	//Flattening sums and matrix
	int* h_sum_starts_x = new int[no_sums];
	int* h_sum_starts_y = new int[no_sums];
	int* h_sum_ends_x = new int[no_sums];
	int* h_sum_ends_y = new int[no_sums];
	int* h_sum_hints = new int[no_sums];
	int* h_sum_lengths = new int[no_sums];
	int* h_sum_dirs = new int[no_sums];


	int* h_perms = new int; // not sure what to do wiht this one here 

	flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

	print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

	int* h_sol_mat;
	h_sol_mat = new int[m * n];
	flatten_sol_mat(sol_mat, h_sol_mat, m, n);

	print_flattened_matrix(h_sol_mat, m, n);

	//Declare device pointers and copy data into device
	int* d_sum_starts_x, * d_sum_starts_y, * d_sum_ends_x, * d_sum_ends_y, * d_sum_hints, * d_sum_lengths, * d_sum_dirs, * d_sol_mat, * d_t_mats;

	int* d_perms;// not sure what to do wiht this one here 

	hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_hints, no_sums * sizeof(int));
	hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
	hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
	hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
	hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim) * sizeof(int)); //Allocating invidual matrix for each GPU thread
	//You may use this array if you will implement a thread-wise solution

	hipMalloc(&d_perms, sizeof(int));// not sure what to do wiht this one here 

	hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_perms, h_perms, sizeof(int), hipMemcpyHostToDevice);// not sure what to do wiht this one here 


	bool* solved = new bool[1];
	*solved = false;
	bool* d_solved;

	hipMalloc(&d_solved, sizeof(bool));
	hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);


	kakuro_kernel << <grid_dim, block_dim >> > (d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
		d_sum_lengths, d_sum_dirs, d_sol_mat, d_perms, d_t_mats, m, n,
		no_sums, d_solved);
	hipDeviceSynchronize();
	//CUDA

	hipMemcpy(h_sol_mat, d_sol_mat, (m * n) * sizeof(int), hipMemcpyDeviceToHost);


	print_flattened_matrix(h_sol_mat, m, n);
	// 
	//TO DO sol_mat_flattened_to_file(mat, d_sol_mat, m, n)
	//Similiar to sol_mat, use hints from mat and values from d_sol_mat

	for (int i = 0; i < n; i++) {
		delete mat[i];
		delete sol_mat[i];
	}

	delete mat;
	delete sol_mat;

	delete h_sum_starts_x;
	delete h_sum_starts_y;
	delete h_sum_ends_x;
	delete h_sum_ends_y;
	delete h_sum_hints;
	delete h_sum_lengths;
	delete h_sum_dirs;
	delete h_sol_mat;

	hipFree(d_t_mats);
	hipFree(d_sum_starts_x);
	hipFree(d_sum_starts_y);
	hipFree(d_sum_ends_x);
	hipFree(d_sum_ends_y);
	hipFree(d_sum_hints);
	hipFree(d_sum_lengths);
	hipFree(d_sum_dirs);
	hipFree(d_sol_mat);


	return 0;
}
